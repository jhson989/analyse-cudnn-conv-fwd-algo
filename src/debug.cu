#include "../conv_fwd_algo.cuh"

std::string MODE_NAME[] = {
    "IMPLICIT_GEMM",
    "IMPLICIT_PRECOMP_GEMM",
    "GEMM",
    "DIRECT",
    "FFT",
    "FFT_TILING",
    "WINOGRAD",
    "WINOGRAD_NONFUSED"
};


/***************************************************************
 * Debug code
 ***************************************************************/
 void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
    if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
        exit(1);
    }
}

void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}

void print_performance(hipEvent_t& start, hipEvent_t& stop, const hipdnnConvolutionFwdAlgo_t& mode, const size_t memory_size) {
    float msec_total=0.0f;
    cudaErrChk( hipEventElapsedTime(&msec_total, start, stop) );

    printf("[%s]\n", MODE_NAME[(int)(mode)].c_str());
    printf(" : %05.3f s , %f GB\n", 1.0f*msec_total/1024.0f, 1.0f*memory_size/1024.0f/1024.0f/1024.0f);

}