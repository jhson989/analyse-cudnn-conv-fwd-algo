#include "../conv_fwd_algo.cuh"


/***************************************************************
 * Debug code
 ***************************************************************/
 void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
    if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
        exit(1);
    }
}

void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}