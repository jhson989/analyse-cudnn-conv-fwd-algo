#include "../conv_fwd_algo.cuh"

std::string MODE_NAME[] = {
    "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_GEMM",
    "HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT",
    "HIPDNN_CONVOLUTION_FWD_ALGO_FFT",
    "HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING",
    "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD",
    "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED"
};


/***************************************************************
 * Debug code
 ***************************************************************/
 void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
    if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
        exit(1);
    }
}

void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}

void print_performance(hipEvent_t& start, hipEvent_t& stop, const hipdnnConvolutionFwdAlgo_t& mode, const size_t memory_size) {
    float msec_total=0.0f;
    cudaErrChk( hipEventElapsedTime(&msec_total, start, stop) );

    std::cout << "MODE : " << MODE_NAME[(int)(mode)] << "\n";
    std::cout << " -- elapsed time : " << msec_total*1e-3 << " s\n";
    std::cout << " -- workspace size : " << memory_size*1e-9 << " GB\n";

}