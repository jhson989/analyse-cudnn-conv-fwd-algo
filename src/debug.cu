#include "../conv_fwd_algo.cuh"


/***************************************************************
 * Debug code
 ***************************************************************/
 void cudnnAssert(hipdnnStatus_t code, const char *file, int line) {
    if (code != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
        exit(1);
    }
}

void cudaAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(1);
    }
}

void print_performance(hipEvent_t& start, hipEvent_t& stop, const hipdnnConvolutionFwdAlgo_t& mode, const size_t memory_size) {
    float msec_total=0.0f;
    cudaErrChk( hipEventElapsedTime(&msec_total, start, stop) );

    std::cout << "MODE : " << mode << "\n";
    std::cout << " -- elapsed time : " << msec_total*1e-3 << " s\n";
    std::cout << " -- workspace size : " << memory_size*1e-9 << " GB\n";

}