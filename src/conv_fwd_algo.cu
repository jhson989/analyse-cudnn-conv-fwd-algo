#include "../conv_fwd_algo.cuh"

    
void launch_conv_fwd(
    /*CUDNN HANDLER*/hipdnnHandle_t& cudnn,
    /*MODE*/hipdnnConvolutionFwdAlgo_t mode,
    /*LAYER CONFIG*/ const int PAD_H, const int PAD_W, const int STRIDE_H, const int STRIDE_W, const int DILATION_H, const int DILATION_W,
    /*INPUT*/ const hipdnnTensorDescriptor_t& input_desc, const float* d_input,
    /*OUTPUT*/ const hipdnnTensorDescriptor_t& output_desc, float* d_output,
    /*FILTER*/ const hipdnnFilterDescriptor_t& filter_desc, const float* d_filter
) {

    /*******************************************************************************
     * Describe convolution forward layer
     ********************************************************************************/ 

    // Layer 
    hipdnnConvolutionDescriptor_t conv2d_desc;
    cudnnErrChk( hipdnnCreateConvolutionDescriptor(&conv2d_desc) );
    cudnnErrChk( hipdnnSetConvolution2dDescriptor(
        conv2d_desc,
        /*PAD_H*/PAD_H, /*PAD_W*/PAD_W, /*STRIDE_VERTICAL*/STRIDE_H, /*STRIDE_HORIZONTAL*/STRIDE_W, /*DILATION_H*/DILATION_H, /*DILATION_W*/DILATION_W, /*MODE*/HIPDNN_CROSS_CORRELATION, /*DATATYPE*/HIPDNN_DATA_FLOAT
    ) );

    // Specify forward algorithm
    void* d_workspace_forward;
    size_t bytes_workspace_forward;
    cudnnErrChk( hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, filter_desc, conv2d_desc, output_desc, mode, &bytes_workspace_forward) );
    cudaErrChk( hipMalloc(&d_workspace_forward, bytes_workspace_forward) );




    /******************************************************************
     * 6. Launch forward kernel
     *******************************************************************/    
    
    hipEvent_t start, stop;
    cudaErrChk( hipEventCreate(&start) );
    cudaErrChk( hipEventCreate(&stop) );

    const float alpha=1.0f, beta=0.0f;
    cudaErrChk( hipEventRecord(start, NULL) );
    for (int i=0; i<TEST_ITERATION; i++) {
        cudnnErrChk( hipdnnConvolutionForward(cudnn
                                            , /*ALPHA*/&alpha
                                            , /*INPUT*/input_desc, d_input
                                            , /*KERNEL*/filter_desc, d_filter
                                            , /*LAYER*/conv2d_desc, mode, d_workspace_forward, bytes_workspace_forward
                                            , /*BETA*/&beta
                                            , /*OUTPUT*/output_desc, d_output
        ) );    
    }
    cudaErrChk( hipDeviceSynchronize() );
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk( hipEventSynchronize(stop) );

    print_performance(start, stop, mode, bytes_workspace_forward);

    cudaErrChk( hipFree(d_workspace_forward) );
}
